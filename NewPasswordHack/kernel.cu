#include "hip/hip_runtime.h"
#include <cmath>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <fstream>

#define POSSIBLE_CHARS 46
#define STARTING_POINT 47
#define THREADS 64//32
#define Z 32
#define XY 32//128
#define OFFSET 512

struct found {
	int yes;
	char password[8];
	unsigned long long attempts;
	unsigned long long lastId;
};
__device__ struct found *foundGlobal; // device (global)
unsigned long hackPassword(const int passwordSize, char actualPassword[8]);

__global__ void addKernel(char actualPassword[8])
{
	if (foundGlobal[0].yes != 1) {
		const unsigned long long idx = (blockIdx.x + blockIdx.y * gridDim.x
			+ gridDim.x * gridDim.y * blockIdx.z) * (blockDim.x * blockDim.y * blockDim.z)
			+ (threadIdx.z * (blockDim.x * blockDim.y))
			+ (threadIdx.y * blockDim.x) + threadIdx.x;

		unsigned long long nextId = foundGlobal[0].lastId;
		bool localGotcha = false;
		for (int i = 0; i < OFFSET && !localGotcha; i++) {
			char answer[8];
			unsigned long long newId = idx*OFFSET + i + nextId;

			int location = 0;
			unsigned long long idx2 = newId;
			
			while (idx2 >= POSSIBLE_CHARS) {
				location++;
				idx2 /= POSSIBLE_CHARS;
			}

			idx2 = newId;
			char pos2;
			while (location > 0) {
				pos2 = (idx2 % POSSIBLE_CHARS) + STARTING_POINT;
				answer[location] = pos2;
				location--;

				idx2 /= POSSIBLE_CHARS;
			}

			answer[0] = idx2 + STARTING_POINT;

			bool right = true;
			for (int z = 0; z < 8 && right; z++) {
				if (answer[z] != actualPassword[z]) {
					right = false;
				}
			}

			if (right) {
				localGotcha = true;
				foundGlobal[0].yes = 1;
				foundGlobal[0].attempts = newId;
				for (int i = 0; i < 8; i++) {
					foundGlobal[0].password[i] = answer[i];
				}

				return;
			}

		}

		if (idx == XY*XY*Z*THREADS - 1) {
			foundGlobal[0].lastId = nextId + (XY*XY*Z*THREADS*OFFSET);
		}
	}
}

int main()
{
	const int repetitions = 1;
	unsigned long times[repetitions];
	unsigned long attempts[repetitions];
	int passwordSize = 8;
	hipError_t cudaStatus;
	char input[8] = { 0 };
	std::cout << "Please enter a password, no greater than 7 characters long, followed by the enter key.\n";
	std::cin.getline(input, 8, '\n');

	for (int i = 0; i < 8; i++) {
		if (input[i] == '\0') {
			passwordSize--;
		}
	}

    // Add vectors in parallel.
	for (int i = 0; i < repetitions; i++) {
		auto begin = std::chrono::high_resolution_clock::now();
		attempts[i] = hackPassword(passwordSize, input);
		auto end = std::chrono::high_resolution_clock::now();
		times[i] = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count();
		hipDeviceSynchronize();
	}

	std::ofstream out("cudaData.txt");
	out << "Time ns, attempts\n";
	for (int i = 0; i < repetitions; i++) {
		out << times[i] << " , " << attempts[i] << "\n";
	}
	out.close();


    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	system("pause");
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
unsigned long hackPassword(int passwordSize, char actualPassword[8])
{
    hipError_t cudaStatus;

	char* d_localactualPassword;
	hipMalloc(&d_localactualPassword, 8 * sizeof(char));
	hipMemcpy(d_localactualPassword, actualPassword, 8 * sizeof(char), hipMemcpyHostToDevice);

	size_t size = 1 * sizeof(struct found);
	size_t sizep = 1 * sizeof(struct found*);
	struct found *localDeviceFound; // device (local)
	hipMalloc(&localDeviceFound, size);
	hipMemcpyToSymbol(HIP_SYMBOL(foundGlobal), &localDeviceFound, sizep);
	struct found *newFound = (struct found*)malloc(size);

	dim3 grid(XY, XY, Z);

	while (newFound[0].yes != 1) {
		// Launch a kernel on the GPU with one thread for each element.
		addKernel << <grid, THREADS >> >(d_localactualPassword);
		hipDeviceSynchronize();
		hipMemcpy(newFound, localDeviceFound, size, hipMemcpyDeviceToHost);	
		if (newFound[0].yes == 1) {
			printf("Found Password, it took: %llu attempts. Password = %s\n\n", newFound[0].attempts, newFound[0].password);
			return newFound[0].attempts;
		}
	}


    // Launch a kernel on the GPU with one thread for each element.

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

Error:
    
    return cudaStatus;
}
